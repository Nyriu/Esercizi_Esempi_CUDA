// file esempio sommavettore_gpu with herror handling

#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 32 // 100
#define NumThPerBlock 32 //256
#define NumBlocks 1

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))


__global__ void add(int *d_a, int *d_b, int *d_c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N)
    d_c[tid] = d_a[tid] + d_b[tid];
}


int main( void ) {
  int a[N], b[N], c[N];     // host variables containing host pointers
  int *dev_a, *dev_b, *dev_c; // host variables containing device pointers

  // static allocation on device memory
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

  // host initializes arrays
  for (int i=0; i<N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  // copy arrays from host to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

  add<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

  // TODO wait threads completion
  //HANDLE_ERROR(cudaDeviceSynchronize());

  //retrieve the result from device dev_c into c
  HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

  //show results
  for (int i=0; i<N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  //free device memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
