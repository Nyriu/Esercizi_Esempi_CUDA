// file esempio tantiprint

#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void miokernel(void){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Sono il thread %d!\n", tid);
}

int main() {
  miokernel<<<2,32>>>();
  printf("Hello World!\n");
  return 0;
}
