// Esercizio
// Come il precedente ma 2D
// Dato array 2D di NxM interi e un valore intero x conta quanti sono gli elementi di uguali a x
//
// x e N forniti da linea di comando // TODO

// Variante
// Usare grid 2D // TODO


#include <hip/hip_runtime.h>
#include <locale>
#include <stdlib.h>
#include <iostream>
#include <experimental/random>
#include <time.h>

#define N 60 // 3 // Rows
#define M 70 // 4 // Cols
#define THREADS_PER_BLOCK 256
#define MAX_VAL 5 // 50

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cout << hipGetErrorString( err ) << " in " << file << " line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))

void init_vec(int v[N][M]) {
  for (int i=0; i<N; i++) {
    for (int j=0; j<M; j++) {
      v[i][j] = std::experimental::randint(0,MAX_VAL);
      //std::cout << "i = " << i << "\tj = " << j << std::endl;
      //std::cout << "v[i][j] = "<< v[i][j] << std::endl;
      //v[i][j] = 0;
      //v[i][j] = i*M + j;
      //std::cout << "v[i][j] = "<< v[i][j] << "\n" << std::endl;
    }
  }
}

void show_vec(int v[N][M]) {
  std::cout << "[\n";
  for (int i=0; i<N; i++) {
    for (int j=0; j<M; j++) {
      std::cout << v[i][j] << ", ";
    }
    std::cout << "\n";
  }
  std::cout << "]\n" << std::endl;
}

int cpu_count(int v[N][M], const int x) {
  int c=0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<M; j++) {
      if (v[i][j] == x)
        c++;
    }
  }
  return c;
}

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}


int compute_num_blocks(const int n, const int m) {
  int b = div_ceil(n*m, THREADS_PER_BLOCK);
  return b;
}


__device__ int d_g_count = 0;
__global__ void gpu_count(int *d_v, const int x) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= N*M)
    return;


  if (d_v[tid] == x) {
    atomicAdd(&d_g_count, 1);
    printf("%d\t%d - Increment\n", tid, d_v[tid]);
  } else {
    printf("%d\t%d\n", tid, d_v[tid]);
  }
}

int main( void ) {
  size_t vec_size = ((size_t)N*M) * sizeof(int);
  //std::cout << "N = " << N << std::endl;
  //std::cout << "sizeof(int) = " << sizeof(int) << std::endl;
  //std::cout << "size = " << vec_size << std::endl;

  int seed = (int)time(NULL);

  //std::experimental::reseed(3);
  std::experimental::reseed(seed);
  std::cout << "seed = " << seed << std::endl;


  int x;
  x = std::experimental::randint(0,MAX_VAL);
  //x = N-1; // x is in the last block


  int v[N][M];
  //int *v;
  //v = (int*)malloc(vec_size);

  init_vec(v);
  show_vec(v);

  int *dev_v;
  HANDLE_ERROR(hipMalloc((void**)&dev_v, vec_size));
  HANDLE_ERROR(hipMemcpy(dev_v, v, vec_size, hipMemcpyHostToDevice));

  int g_count=0;
  int b = compute_num_blocks(N,M);
  std::cout <<
    "\nN = " << N <<
    "\nM = " << M <<
    "\nN*M = " << N*M <<
    "\nthreads = " << THREADS_PER_BLOCK <<
    "\nb = " << b <<
    "\nb*threads = " << b*THREADS_PER_BLOCK << "\n" <<
    std::endl;

  gpu_count<<<b, THREADS_PER_BLOCK>>>(dev_v, x);
  HANDLE_ERROR(hipDeviceSynchronize());

  HANDLE_ERROR(hipMemcpyFromSymbol(&g_count, HIP_SYMBOL(d_g_count), sizeof(int))); // better than cudaMemcpy // Look at ref in README

  int c_count = cpu_count(v,x);

  std::cout << "\nx = " << x << "\tMAX_VAL = " << MAX_VAL << "\n" << std::endl;
  std::cout << "c_count = " << c_count << std::endl;
  std::cout << "g_count = " << g_count << std::endl;

  if (c_count == g_count)
    std::cout << "\nCorrect" << std::endl;
  else
    std::cout << "\nWRONG!" << std::endl;

  hipFree(&d_g_count);
  hipFree(dev_v);
  //free(v);

  return 0;
}
