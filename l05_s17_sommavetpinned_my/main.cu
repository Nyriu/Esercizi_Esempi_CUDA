// vector sum with pinned memory (page-locked)

#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 100 // 10 // 32 // 100
#define NumThPerBlock 64 // 32 //256
#define NumBlocks 2 // 1

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))


__global__ void add(int *d_a, int *d_b, int *d_c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N)
    d_c[tid] = d_a[tid] + d_b[tid];
  else
    printf("I'm a padding thread tid=%d\n", tid);
}


int main( void ) {
  int *a, *b, *c;     // host variables containing host pointers
  int *dev_a, *dev_b, *dev_c; // host variables containing device pointers

  // pinned memory on host
  hipSetDeviceFlags(hipDeviceMapHost);
  HANDLE_ERROR(hipHostAlloc(&a, N*sizeof(int), hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc(&b, N*sizeof(int), hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc(&c, N*sizeof(int), hipHostMallocMapped));

  // static allocation on device memory
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

  // host initializes arrays
  for (int i=0; i<N; i++) {
    a[i] = -i;
    b[i] = i * i;
    c[i] = 0;
  }

  // copy arrays from host to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

  add<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

  // wait threads completion
  HANDLE_ERROR(hipDeviceSynchronize());

  //retrieve the result from device dev_c into c
  HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

  //show results
  for (int i=0; i<N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  //free device memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
