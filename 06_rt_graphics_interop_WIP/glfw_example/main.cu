#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES

#include <iostream>
#include <stdlib.h>

#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cout << "Error Name: " << hipGetErrorName( err ) << std::endl;
    std::cout << hipGetErrorString( err ) << " in " << file << " line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))


static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}


// Globals
#define DIM 512

GLuint bufferObj;
hipGraphicsResource *resource;


__global__ void kernel(uchar4 *ptr) {
  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  // now calculate the value at that position
  float fx = x/(float)DIM - 0.5f;
  float fy = y/(float)DIM - 0.5f;
  unsigned char green =
    128 + 127 * sin(abs(fx*100) - abs(fy*100));

  // accessing uchar4 vs unsigned char*
  ptr[offset].x = 0;
  ptr[offset].y = green;
  ptr[offset].z = 0;
  ptr[offset].w = 255;
}


int main(int argc, char **argv) {
  hipDeviceProp_t prop;
  int dev;

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 0;
  HANDLE_ERROR(
      hipChooseDevice(&dev,&prop)
      );
  //HANDLE_ERROR(
  //    cudaGLSetGLDevice(dev) // deprecated
  //    );


	if (!glfwInit()) exit(EXIT_FAILURE);
	if (atexit(glfwTerminate)) {
		glfwTerminate();
		exit(EXIT_FAILURE);
	}

	GLFWwindow* window;
	window = glfwCreateWindow(DIM, DIM, "Interop Example (GLFW)", NULL, NULL);
	if (!window) exit(EXIT_FAILURE);

  glfwSetKeyCallback(window, key_callback);

	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);


  std::cout << glGetString(GL_VENDOR) << std::endl;
  std::cout << glGetString(GL_RENDERER) << std::endl;

  // TODO ARB or not ARB
	//glGenBuffers(1, &pbo);
	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	//glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * sizeof(GLubyte)*WIDTH*HEIGHT, NULL, GL_DYNAMIC_DRAW);
  glGenBuffers(1, &bufferObj);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM*DIM*4, NULL, GL_DYNAMIC_DRAW_ARB);

  HANDLE_ERROR(
      hipGraphicsGLRegisterBuffer(
        &resource,
        bufferObj,
        cudaGraphicsMapFlagsNone
        )
      );

  uchar4* devPtr;
  size_t size;
  HANDLE_ERROR(
      hipGraphicsMapResources(1, &resource, NULL)
      );
  HANDLE_ERROR(
      hipGraphicsResourceGetMappedPointer(
        (void**)&devPtr,
        &size,
        resource
        )
      );

  dim3 grids(DIM/16, DIM/16);
  dim3 threads(16,16);
  kernel<<<grids,threads>>>(devPtr);

  HANDLE_ERROR(
      hipGraphicsUnmapResources(1, &resource, NULL)
      );

	while (!glfwWindowShouldClose(window)) {
		//kernelUpdate(WIDTH, HEIGHT);
		glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);
		glfwSwapBuffers(window);

    // Poll for and process events
    glfwPollEvents();
	}



  HANDLE_ERROR(
      hipGraphicsUnregisterResource(resource)
      );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0);
  glDeleteBuffers(1, &bufferObj);

  glfwTerminate();

  return 0;
}

