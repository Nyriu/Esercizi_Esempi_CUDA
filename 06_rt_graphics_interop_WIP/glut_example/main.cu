#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES

#include <iostream>
#include <stdlib.h>

#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cout << "Error Name: " << hipGetErrorName( err ) << std::endl;
    std::cout << hipGetErrorString( err ) << " in " << file << " line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))



// Globals
#define DIM 512

GLuint bufferObj;
hipGraphicsResource *resource;



__global__ void kernel(uchar4 *ptr) {
  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  // now calculate the value at that position
  float fx = x/(float)DIM - 0.5f;
  float fy = y/(float)DIM - 0.5f;
  unsigned char green =
    128 + 127 * sin(abs(fx*100) - abs(fy*100));

  // accessing uchar4 vs unsigned char*
  ptr[offset].x = 0;
  ptr[offset].y = green;
  ptr[offset].z = 0;
  ptr[offset].w = 255;
}

static void draw_func( void ){
  glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);
  glutSwapBuffers();
}

static void key_func(unsigned char key, int x, int y){
  switch(key) {
    case 27: // ESC
      // clean OpenGL and CUDA
      HANDLE_ERROR(
          hipGraphicsUnregisterResource(resource)
          );
      glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0);
      glDeleteBuffers(1, &bufferObj);
      exit(0);
  }
}











int main(int argc, char **argv) {
  hipDeviceProp_t prop;
  int dev;

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 0;
  HANDLE_ERROR(
      hipChooseDevice(&dev,&prop)
      );
  //HANDLE_ERROR(
  //    cudaGLSetGLDevice(dev) // deprecated
  //    );

  // these GLUT calls need to be made before the other GL calls
  glutInit(&argc, argv);
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
  glutInitWindowSize( DIM, DIM );
  glutCreateWindow( "Interop Example" );

  glGenBuffers(1, &bufferObj);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM*DIM*4, NULL, GL_DYNAMIC_DRAW_ARB);

  HANDLE_ERROR(
      hipGraphicsGLRegisterBuffer(
        &resource,
        bufferObj,
        cudaGraphicsMapFlagsNone
        )
      );

  uchar4* devPtr;
  size_t size;
  HANDLE_ERROR(
      hipGraphicsMapResources(1, &resource, NULL)
      );
  HANDLE_ERROR(
      hipGraphicsResourceGetMappedPointer(
        (void**)&devPtr,
        &size,
        resource
        )
      );

  dim3 grids(DIM/16, DIM/16);
  dim3 threads(16,16);
  kernel<<<grids,threads>>>(devPtr);

  HANDLE_ERROR(
      hipGraphicsUnmapResources(1, &resource, NULL)
      );

  // set up GLUT and kick off main loop
  glutKeyboardFunc(key_func);
  glutDisplayFunc(draw_func);
  glutMainLoop();

  return 0;
}

