#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
//#include <hip/hip_cooperative_groups.h>
//using namespace cooperative_groups;
//// Alternatively use an alias to avoid polluting the namespace with collective algorithms
////namespace cg = cooperative_groups;

////for __syncthreads()
//#ifndef __HIPCC__ 
//#define __HIPCC__
//#endif
//#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_cooperative_groups.h>



static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cout << "Error Name: " << hipGetErrorName( err ) << std::endl;
    std::cout << hipGetErrorString( err ) << " in " << file << " line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))


enum PolygonType { none, rect, triang };
struct PolygonInfo {
  public:
    int width, height;
    PolygonType ptype;
    __host__ __device__ PolygonInfo(int w, int h, PolygonType t) :
      width(w), height(h), ptype(t) {}
};
class Polygon {
  protected:
    int width, height;
  public:
    __host__ __device__ Polygon(int w, int h) : width(w), height(h) {}
    __host__ __device__ Polygon(const PolygonInfo& pi) : width(pi.width), height(pi.height) {}
    __host__ __device__  void set_values(int a, int b) {
      width=a;
      height=b;
    }
    __host__ __device__  virtual int area() {
      printf("\nLOL here!\n");
      printf(" width = %d\n height = %d\n", width, height);
      return 0;
    }
    __host__ __device__  virtual PolygonInfo get_info() {
      return PolygonInfo(width,height,PolygonType::none);
    }
};

class Rectangle: public Polygon {
  public:
    __host__ __device__ Rectangle(int w, int h) : Polygon(w,h) {}
    __host__ __device__ Rectangle(const PolygonInfo& pi) : Polygon(pi) {}
    __host__ __device__  int area() override {
      return width * height;
    }
    __host__ __device__  PolygonInfo get_info() override {
      printf("rect get_info\n");
      return PolygonInfo(width,height,PolygonType::rect);
    }
};

class Triangle: public Polygon {
  public:
    __host__ __device__ Triangle(int w, int h) : Polygon(w,h) {}
    __host__ __device__ Triangle(const PolygonInfo& pi) : Polygon(pi) {}
    __host__ __device__   int area() override {
      return (width * height / 2);
    }
    __host__ __device__  PolygonInfo get_info() override {
      printf("triang get_info\n");
      return PolygonInfo(width,height,PolygonType::triang);
    }
};



static __global__ void wrong_example_kernel(Polygon *pols, int n_pols) {
  Polygon *p = pols;
  for (int i=0; i<n_pols; i++) {
    printf("(device) p->area() = %d", p->area());
    p++;
  }
}


static __global__ void inst_obj_dev_kernel(PolygonInfo *pols_infos, int n_pols) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  //printf("%d %d\n", x,y);

  size_t pols_size = sizeof(Polygon)*n_pols;
  Polygon *pols = (Polygon*) malloc(pols_size);

  for (int i=0; i<n_pols; i++) {
    Polygon *tmp_p = nullptr;
    //PolygonInfo tmp_pi = *(pols_infos+i);
    PolygonInfo *pi_p = pols_infos+i;

    //printf("w=%d, h=%d, ptype=%d\n", pi_p->width, pi_p->height, pi_p->ptype);

    if (pi_p->ptype == PolygonType::rect) {
      tmp_p = new Rectangle(*(pols_infos+i));
    } else if (pi_p->ptype == PolygonType::triang) {
      tmp_p = new Triangle(*(pols_infos+i));
    } else if (pi_p->ptype == PolygonType::none) {
      tmp_p = new Polygon(*(pols_infos+i));
    } else {
      printf("we have a problem...\n");
    }

    memcpy(pols+i, tmp_p, sizeof(*tmp_p));
  }

  printf("%d %d\n", x,y);
  for (int i=0; i<n_pols; i++) {
    printf("\t area = %d\n", pols[i].area());
  }

}


__device__ Polygon *pols = nullptr;
static __global__ void kernel(PolygonInfo *pols_infos, int n_pols) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  //printf("%d %d\n", x,y);

  if (x == 0 && y == 0) {
    printf("%d %d\n I'm instancing...\n", x,y);
    size_t pols_size = sizeof(Polygon)*n_pols;
    pols = (Polygon*) malloc(pols_size);

    for (int i=0; i<n_pols; i++) {
      Polygon *tmp_p = nullptr;
      //PolygonInfo tmp_pi = *(pols_infos+i);
      PolygonInfo *pi_p = pols_infos+i;

      //printf("w=%d, h=%d, ptype=%d\n", pi_p->width, pi_p->height, pi_p->ptype);

      if (pi_p->ptype == PolygonType::rect) {
        tmp_p = new Rectangle(*(pols_infos+i));
      } else if (pi_p->ptype == PolygonType::triang) {
        tmp_p = new Triangle(*(pols_infos+i));
      } else if (pi_p->ptype == PolygonType::none) {
        tmp_p = new Polygon(*(pols_infos+i));
      } else {
        printf("we have a problem...\n");
      }
      memcpy(pols+i, tmp_p, sizeof(*tmp_p));
    }
  }

  __syncthreads(); // this is needed
  // this handle only the case of one block

  if (x == 0 && y == 0) {
    for (int i=0; i<n_pols; i++) {
      printf("%d %d\n area = %d\n", x, y, pols[i].area());
    }
  } else {
    //printf("%d %d\n I do nothig atm...\n pols = %p\n", x,y, pols);
    for (int i=0; i<n_pols; i++) {
      printf("%d %d\n pols=%p, area = %d\n", x, y, pols, pols[i].area());
    }
  }

}


int main() {
  hipDeviceProp_t prop;
  int dev;
  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 0;
  HANDLE_ERROR(
      hipChooseDevice(&dev,&prop)
      );

  // Host init
  //Polygon tri = Triangle(3,4);
  Triangle tri(3,4);
  std::cout << "tri.area() = " << tri.area() << std::endl;
  Rectangle rec(5,7);
  std::cout << "rec.area() = " << rec.area() << std::endl;

  std::vector<Polygon*> pols;
  pols.push_back(&tri);
  pols.push_back(&rec);

  for (Polygon *p : pols) {
    std::cout << "p->area() = " << p->area() << std::endl;
  }

  // Now I want to move the vector to GPU and
  // for each elem call area() from device


  // { /// WRONG WAY
  //   size_t total_size = 0;
  //   for (Polygon *p : pols) {
  //     total_size += sizeof(*p);
  //   }

  //   Polygon *dev_pols = nullptr;
  //   HANDLE_ERROR(
  //       hipMalloc((void**)&dev_pols, total_size)
  //       );

  //   int offset = 0;
  //   for (Polygon *p : pols) {
  //     HANDLE_ERROR(
  //         hipMemcpy((void*)(dev_pols+offset), (void*)p, sizeof(*p), hipMemcpyHostToDevice)
  //         );
  //     offset++;
  //   }
  //   float grids = 1;
  //   float threads = 1;
  //   wrong_example_kernel<<<grids,threads>>>(dev_pols, pols.size());
  //   // this generate a wrong mem access because vtable on host
  // } /// END // WRONG WAY

  //{ /// INSTACING OBJS ON DEVICE
  //  size_t pols_infos_size = sizeof(PolygonInfo)*pols.size();
  //  PolygonInfo *pols_infos = (PolygonInfo *) malloc(pols_infos_size);
  //  int i = 0;
  //  for (Polygon *p : pols) {
  //    PolygonInfo pi = p->get_info();
  //    memcpy(&pols_infos[i], &pi, sizeof(pi));
  //    i++;
  //  }

  //  PolygonInfo *dev_pols_infos = nullptr;
  //  HANDLE_ERROR(
  //      hipMalloc((void**)&dev_pols_infos, pols_infos_size)
  //      );
  //  HANDLE_ERROR(
  //      hipMemcpy((void*)dev_pols_infos, (void*)pols_infos, pols_infos_size, hipMemcpyHostToDevice)
  //      );

  //  free(pols_infos);

  //  dim3 grids(1);
  //  dim3 threads(1);
  //  inst_obj_dev_kernel<<<grids,threads>>>(dev_pols_infos, pols.size());
  //} /// END // INSTACING OBJS ON DEVICE


  { /// AS BEFORE BUT ONE THREAD INST AND THE OTHER USE
    size_t pols_infos_size = sizeof(PolygonInfo)*pols.size();
    PolygonInfo *pols_infos = (PolygonInfo *) malloc(pols_infos_size);
    int i = 0;
    for (Polygon *p : pols) {
      PolygonInfo pi = p->get_info();
      memcpy(&pols_infos[i], &pi, sizeof(pi));
      i++;
    }

    PolygonInfo *dev_pols_infos = nullptr;
    HANDLE_ERROR(
        hipMalloc((void**)&dev_pols_infos, pols_infos_size)
        );
    HANDLE_ERROR(
        hipMemcpy((void*)dev_pols_infos, (void*)pols_infos, pols_infos_size, hipMemcpyHostToDevice)
        );

    free(pols_infos);

    dim3 grids(1);
    dim3 threads(10,10);
    kernel<<<grids,threads>>>(dev_pols_infos, pols.size());
  } /// END

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipDeviceReset());

  return 0;
}

