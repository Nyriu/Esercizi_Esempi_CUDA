#include <iostream>
#include <hip/hip_runtime.h>

static void HandleError( hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cout << "Error Name: " << hipGetErrorName( err ) << std::endl;
    std::cout << hipGetErrorString( err ) << " in " << file << " line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))


class Base {
  public:
    __device__ virtual void fun1() const  {
    //__device__ void fun1() const  {
      printf("Base fun1\n");
    }

    __device__ void fun2() const {
      printf("Base fun2\n");
      fun1();
    }
};

class Derived : public Base {
  public:
    __device__ void fun1() const override {
    //__device__ void fun1() const {
      printf("Derived fun2\n");
    }
};



static __global__ void kernel() {
  //Base *obj = new Base();
  Base *obj = new Derived();
  printf("inside kernel\n");
  obj->fun1();
  obj->fun2();
}


int main() {
  hipDeviceProp_t prop;
  int dev;
  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 0;
  HANDLE_ERROR(
      hipChooseDevice(&dev,&prop)
      );

  float grids = 1;
  float threads = 1;
  kernel<<<grids,threads>>>();

  HANDLE_ERROR(hipDeviceSynchronize());

  return 0;
}

