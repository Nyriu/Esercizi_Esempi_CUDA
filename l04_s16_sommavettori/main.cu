// file esempio sommavettore_gpu

#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 32 // 100
#define NumThPerBlock 32 //256
#define NumBlocks 1

__global__ void add(int *d_a, int *d_b, int *d_c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N)
    d_c[tid] = d_a[tid] + d_b[tid];
}



int main( void ) {
  int a[N], b[N], c[N];     // host variables containing host pointers
  int *dev_a, *dev_b, *dev_c; // host variables containing device pointers

  // static allocation on device memory
  hipMalloc((void**)&dev_a, N*sizeof(int));
  hipMalloc((void**)&dev_b, N*sizeof(int));
  hipMalloc((void**)&dev_c, N*sizeof(int));

  // host initializes arrays
  for (int i=0; i<N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  // copy arrays from host to device
  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<NumBlocks, NumThPerBlock>>>(dev_a, dev_b, dev_c);

  // TODO wait threads completion

  //retrieve the result from device dev_c into c
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  //show results
  for (int i=0; i<N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  //free device memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
