#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#define WIDTH 512
#define HEIGHT 512

extern "C" void kernelBindPbo(GLuint pixelBufferObj);
extern "C" void kernelUpdate(int width, int height);
extern "C" void kernelExit(GLuint pixelBufferObj);

GLuint pbo;

int main() {
	if (!glfwInit()) exit(EXIT_FAILURE);
	if (atexit(glfwTerminate)) {
		glfwTerminate();
		exit(EXIT_FAILURE);
	}

	GLFWwindow* window;
	window = glfwCreateWindow(WIDTH, HEIGHT, "gl-cuda-test", NULL, NULL);
	if (!window) exit(EXIT_FAILURE);

	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);
	if (glewInit() != GLEW_OK) exit(EXIT_FAILURE);

  std::cout << glGetString(GL_VENDOR) << std::endl;
  std::cout << glGetString(GL_RENDERER) << std::endl;

	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * sizeof(GLubyte)*WIDTH*HEIGHT, NULL, GL_DYNAMIC_DRAW);

	kernelBindPbo(pbo);

	while (!glfwWindowShouldClose(window)) {
		kernelUpdate(WIDTH, HEIGHT);
		glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
		glfwSwapBuffers(window);
	}

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	kernelExit(pbo);
	glDeleteBuffers(1, &pbo);

	//getchar();

	return 0;
}

#include <iostream>

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

hipGraphicsResource *cudapbo;

extern "C" void kernelBindPbo(GLuint pixelBufferObj) {
	gpuErrchk(hipGraphicsGLRegisterBuffer(&cudapbo, pixelBufferObj, hipGraphicsRegisterFlagsWriteDiscard));
}

extern "C" void kernelExit(GLuint pixelBufferObj) {
	gpuErrchk(cudaGLUnregisterBufferObject(pixelBufferObj));
	gpuErrchk(hipGraphicsUnregisterResource(cudapbo));
}

__global__ void kernel(uchar4 *map, unsigned char frame) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = x + y * blockDim.x * gridDim.x;

	map[id].x = x / 2;
	map[id].y = y / 2;
	map[id].z = frame;
	map[id].w = 255;
}

extern "C" void kernelUpdate(int width, int height) {
	static unsigned char frame = 0;
	frame++;
	uchar4 *dev_map;

	gpuErrchk(hipGraphicsMapResources(1, &cudapbo, NULL));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&dev_map, NULL, cudapbo));

	dim3 threads(8, 8);
	dim3 grids(width / 8, height / 8);
	kernel << <grids, threads >> > (dev_map, frame);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipGraphicsUnmapResources(1, &cudapbo, NULL));
}
